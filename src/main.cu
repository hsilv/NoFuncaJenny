#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <fstream>
#include <algorithm>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include "pgm.h"

const double degreeInc = 0.5;
const int degreeBins = static_cast<int>(180 / degreeInc);
const int rBins = 100;
const double radInc = degreeInc * M_PI / 180;

// The CPU function returns a pointer to the accummulator
void CPU_HoughTran (unsigned char *pic, int w, int h, int **acc)
{
    double rMax = sqrt (1.0 * w * w + 1.0 * h * h) / 2.0;  //(w^2 + h^2)/2, radio max equivalente a centro -> esquina
    *acc = new int[rBins * degreeBins];            //el acumulador, conteo depixeles encontrados, 90*180/degInc = 9000
    memset (*acc, 0, sizeof(int) * rBins * degreeBins); //init en ceros
    int xCent = w / 2;
    int yCent = h / 2;
    double rScale = 2.0 * rMax / rBins;

    for (int i = 0; i < w; i++) //por cada pixel
        for (int j = 0; j < h; j++) //...
        {
            int idx = j * w + i;
            if (pic[idx] > 0) //si pasa thresh, entonces lo marca
            {
                int xCoord = i - xCent;
                int yCoord = yCent - j;  // y-coord has to be reversed
                double theta = 0.0;         // actual angle
                for (int tIdx = 0; tIdx < degreeBins; tIdx++) //add 1 to all lines in that pixel
                {
                    double r = xCoord * cos (theta) + yCoord * sin (theta);
                    int rIdx = (r + rMax) / rScale;
                    if (rIdx >= 0 && rIdx < rBins)
                    {
                        (*acc)[rIdx * degreeBins + tIdx]++; //+1 para este radio r y este theta
                    }
                    theta += radInc;
                }
            }
        }
}

// GPU kernel. One thread per image pixel is spawned.
__global__ void GPU_HoughTran(unsigned char *pic, int w, int h, int *acc, double rMax, double rScale, double *d_Cos, double *d_Sin) {
    int gloID = blockIdx.x * blockDim.x + threadIdx.x;
    if (gloID >= w * h) return;  // Limitar el acceso a hilos válidos

    int xCent = w / 2;
    int yCent = h / 2;

    int xCoord = (gloID % w) - xCent;
    int yCoord = yCent - (gloID / w);

    if (pic[gloID] > 0) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            double r = xCoord * d_Cos[tIdx] + yCoord * d_Sin[tIdx];
            int rIdx = (r + rMax) / rScale;
            if (rIdx >= 0 && rIdx < rBins)
            {
            atomicAdd(acc + (rIdx * degreeBins + tIdx), 1);
            }
        }
    }
}

int main(int argc, char **argv) {
    int i;

    if (argc < 2) {
        printf("Usage: %s <input_image.pgm>\n", argv[0]);
        return -1;
    }

    PGMImage inImg(argv[1]);
    int *cpuht;
    int w = inImg.x_dim;
    int h = inImg.y_dim;
    
    double* d_Cos;
    double* d_Sin;

    hipMalloc ((void **) &d_Cos, sizeof (double) * degreeBins);
    hipMalloc ((void **) &d_Sin, sizeof (double) * degreeBins);

    // CPU calculation
    CPU_HoughTran(inImg.pixels, w, h, &cpuht);

    double *pcCos = (double *) malloc(sizeof(double) * degreeBins);
    double *pcSin = (double *) malloc(sizeof(double) * degreeBins);
    double rad = 0;
    for (int i = 0; i < degreeBins; i++) {
        pcCos[i] = cos(rad);
        pcSin[i] = sin(rad);
        rad += radInc;
    }

    hipMemcpy(d_Cos, pcCos, sizeof(double) * degreeBins, hipMemcpyHostToDevice);
    hipMemcpy(d_Sin, pcSin, sizeof(double) * degreeBins, hipMemcpyHostToDevice);

    double rMax = sqrt(1.0 * w * w + 1.0 * h * h) / 2;
    double rScale = 2 * rMax / rBins;

    unsigned char *d_in, *h_in;
    int *d_hough, *h_hough;

    h_in = inImg.pixels; // h_in contiene los pixeles de la imagen

    h_hough = (int *) malloc (degreeBins * rBins * sizeof (int));

    hipMalloc ((void **) &d_in, sizeof (unsigned char) * w * h);
    hipMalloc ((void **) &d_hough, sizeof (int) * degreeBins * rBins);
    hipMemcpy (d_in, h_in, sizeof (unsigned char) * w * h, hipMemcpyHostToDevice);
    hipMemset (d_hough, 0, sizeof (int) * degreeBins * rBins);

    // execution configuration uses a 1-D grid of 1-D blocks, each made of 256 threads
    //1 thread por pixel
    int blockNum = ceil (w * h / 256);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    GPU_HoughTran<<<blockNum, 256>>>(d_in, w, h, d_hough, rMax, rScale, d_Cos, d_Sin);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel Execution Time: %f ms\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_hough, d_hough, sizeof(int) * degreeBins * rBins, hipMemcpyDeviceToHost);

    for (i = 0; i < degreeBins * rBins; i++)
    {
        if (cpuht[i] != h_hough[i])
        printf ("Calculation mismatch at : %i %i %i\n", i, cpuht[i], h_hough[i]);
    }
    printf("Done!\n");


    const int threshold = 4240 ;

    // Crear imagen a color para dibujar las líneas
    cv::Mat img(h, w, CV_8UC1, inImg.pixels);
    cv::Mat imgColor;
    cvtColor(img, imgColor, cv::COLOR_GRAY2BGR);

    int xCenter = (w / 2);
    int yCenter = (h / 2);

    // Vector para almacenar líneas con sus pesos
    std::vector<std::pair<cv::Vec2f, int>> linesWithWeights;

    // Recorrer el acumulador y recoger las líneas
    for (int rIdx = 0; rIdx < rBins; rIdx++) {
        for (int tIdx = 0; tIdx < degreeBins; tIdx++) {
            int weight = h_hough[(rIdx * degreeBins) + tIdx];

            if (weight > threshold) {  // Solo incluir líneas relevantes
                float rValue = ((rIdx * rScale) - rMax);  // Distancia r
                float theta = (tIdx * radInc);           // Ángulo theta
                linesWithWeights.push_back(std::make_pair(cv::Vec2f(theta, rValue), weight));
            }
        }
    }

    // Ordenar las líneas por peso en orden descendente
    std::sort(
        linesWithWeights.begin(),
        linesWithWeights.end(),
        [](const std::pair<cv::Vec2f, int> &a, const std::pair<cv::Vec2f, int> &b) {
            return a.second > b.second;
        }
    );

    // Dibujar las líneas principales
    for (int i = 0; i < std::min(threshold, static_cast<int>(linesWithWeights.size())); i++) {
        cv::Vec2f lineParams = linesWithWeights[i].first;
        float theta = lineParams[0];
        float r = lineParams[1];

        double cosTheta = cos(theta);
        double sinTheta = sin(theta);

        double xValue = xCenter - (r * cosTheta);
        double yValue = yCenter - (r * sinTheta);
        double alpha = 1000;  // Factor para extender las líneas

        // Dibujar la línea
        cv::line(
            imgColor,
            cv::Point(cvRound(xValue + (alpha * (-sinTheta))),
                    cvRound(yValue + (alpha * cosTheta))),
            cv::Point(cvRound(xValue - (alpha * (-sinTheta))),
                    cvRound(yValue - (alpha * cosTheta))),
            cv::Scalar(0, 185, 0),
            1,
            cv::LINE_AA
        );
    }

    // Guardar la imagen resultante
    cv::imwrite("outputGlobal.png", imgColor);
    printf("Generated marked image: output.png \n");

    hipFree(d_in);
    hipFree(d_hough);
    hipFree(d_Cos);
    hipFree(d_Sin);
    free(h_hough);
    free(pcCos);
    free(pcSin);

    return 0;
}